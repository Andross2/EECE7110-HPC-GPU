
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>


__global__ void
vec_add_kernel (int *a, int *b, int *c, int len)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  while (tid < len)
    {
      c[tid] = a[tid] + b[tid];
      tid += blockDim.x * gridDim.x;
    }
}


int
main (int args, char **argv)
{
  int len = 1024;
  int *a = new int[len];
  int *b = new int[len];
  int *c = new int[len];
  for (int i = 0; i < len; i++)
    {
      a[i] = rand () % 1024;
      b[i] = rand () % 512;
    }

  int *a_d, *b_d, *c_d;

  hipMalloc ((void **) &a_d, sizeof (int) * len);
  hipMalloc ((void **) &b_d, sizeof (int) * len);
  hipMalloc ((void **) &c_d, sizeof (int) * len);

  hipMemcpy (a_d, a, sizeof (int) * len, hipMemcpyHostToDevice);
  hipMemcpy (b_d, b, sizeof (int) * len, hipMemcpyHostToDevice);
  vec_add_kernel <<< 256, 256 >>> (a_d, b_d, c_d, len);

  hipMemcpy (c, c_d, sizeof (int) * len, hipMemcpyDeviceToHost);

  for (int i = 0; i < len; i++)
    assert ((a[i] + b[i]) == c[i]);

  printf ("Succeed!\n");
  return 0;
}
