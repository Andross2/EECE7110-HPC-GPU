#include "hip/hip_runtime.h"
#include "../helper/util.h"

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>


__global__ void
vec_add_kernel (int *a, int *b, int *c, int len)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  while (tid < len)
    {
      c[tid] = a[tid] + b[tid];
      tid += blockDim.x * gridDim.x;
    }
}


int
main (int args, char **argv)
{
  int len = (2<<28);
  int *a = new int[len];
  int *b = new int[len];
  int *c = new int[len];
  for (int i = 0; i < len; i++)
    {
      a[i] = rand () % 1024;
      b[i] = rand () % 512;
    }

  int *a_d, *b_d, *c_d;

  H_ERR(hipMalloc ((void **) &a_d, sizeof (int) * len));
  H_ERR(hipMalloc ((void **) &b_d, sizeof (int) * len));
  H_ERR(hipMalloc ((void **) &c_d, sizeof (int) * len));

  H_ERR(hipMemcpy (a_d, a, sizeof (int) * len, hipMemcpyHostToDevice));
  H_ERR(hipMemcpy (b_d, b, sizeof (int) * len, hipMemcpyHostToDevice));
  vec_add_kernel <<< 256, 256 >>> (a_d, b_d, c_d, len);

  H_ERR(hipMemcpy (c, c_d, sizeof (int) * len, hipMemcpyDeviceToHost));

  for (int i = 0; i < len; i++)
    assert ((a[i] + b[i]) == c[i]);

  printf ("Succeed!\n");
  return 0;
}
