
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void normal(int *a, int *b, int *c, int len)
{
	int myrow = blockIdx.x;
	__shared__ int smem[256];

	while(myrow <len)
	{
		for (int i = 0; i < len; i ++)//which col of right matrix
		{
			int tid = threadIdx.x;
			int res = 0;
			while( tid < len ) // vector vector multiplication
			{
				res += a[myrow*len + tid] * b[tid*len + i];
				tid += blockDim.x;
			}

			smem[threadIdx.x] = res;
			__syncthreads();
			for (int idx = blockDim.x/2; idx > 0; idx = idx/2)
			{
				if(threadIdx.x < idx)
				{
					int temp = smem[threadIdx.x] + smem[threadIdx.x + idx];
					smem[threadIdx.x] = temp;
				}
				__syncthreads();

			}

			c[myrow*len + i] = smem[0];
		}
	}

	myrow += gridDim.x;

}

__global__ void transpose(int *a, int *b, int *c, int len)
{
	int myrow = blockIdx.x;
	__shared__ int smem[256];

	while(myrow <len)
	{
		for (int i = 0; i < len; i ++)//which col of right matrix
		{
			int tid = threadIdx.x;
			int res = 0;
			while( tid < len ) // vector vector multiplication
			{
				res += a[myrow*len + tid] * b[i*len + tid];
				tid += blockDim.x;
			}
			smem[threadIdx.x] = res;
			__syncthreads();
			for (int i = blockDim.x/2; i > 0; i = i/2)
			{
				if(threadIdx.x < i)
				{
					int temp = smem[threadIdx.x] + smem[threadIdx.x + i];
					smem[threadIdx.x] = temp;
				}
				__syncthreads();

			}

			c[myrow*len + i] = smem[0];
		}

		myrow += gridDim.x;
	}
}
