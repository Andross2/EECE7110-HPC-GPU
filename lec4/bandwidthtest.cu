#include <iostream>
#define N (1<<23)
#include "../helper/wtime.h"

int main(int args, char **argv)
{
    int *a, *a_d;

    hipMalloc((void **)&a_d, sizeof(int)*N);

    a = new int[N];

    double time = wtime();
    hipMemcpy(a_d, a, sizeof(int)*N, hipMemcpyHostToDevice);
    time = wtime() - time;

    std::cout<<"Bandwidth: "<<((N*4)>>20)/time<<" MB/s\n";

    return 0;
}
