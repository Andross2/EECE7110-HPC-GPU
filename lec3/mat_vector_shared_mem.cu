#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include "../helper/util.h"
#include "../helper/wtime.h"
#include <assert.h>

	__global__ void
gpu_mat_vec_multiply(int *mat, int *vec, int *res, int num_rows, int num_cols)
{

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < num_rows)
	{
		int temp_res = 0;
		for (int j = 0; j < num_cols; j ++)
		{
			temp_res += mat[tid * num_cols + j] * vec[j];
		}

		res[tid] = temp_res;
		tid += blockDim.x * gridDim.x;
	}
}

	__global__ void
gpu_mat_vec_multiply_shared(int *mat, int *vec, int *res, int num_rows, int num_cols)
{
    __shared__ int smem[256];
    
    assert(num_cols<=256);
    smem[threadIdx.x] = vec[threadIdx.x];

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < num_rows)
	{
		int temp_res = 0;
		for (int j = 0; j < num_cols; j ++)
		{
			temp_res += mat[tid * num_cols + j] * smem[j];
		}

		res[tid] = temp_res;
		tid += blockDim.x * gridDim.x;
	}
}



	void
mat_vec_multiply(int *mat, int *vec, int *res, int num_rows, int num_cols)
{
	for(int i = 0; i < num_rows; i ++)
	{
		int temp_res = 0;
		for (int j = 0; j < num_cols; j ++)
		{
			temp_res += mat[i * num_cols + j] * vec[j];
		}

		res[i] = temp_res;
	}
}

int main (int args, char **argv)
{
	int num_rows = 256;
	int num_cols = 256;

	int *mat = (int *)malloc(sizeof(int) * num_rows * num_cols);
	int *vec = (int *)malloc(sizeof(int) * num_cols);
	int *res = (int *)malloc(sizeof(int) * num_rows);


	//init matrix
	for (int i = 0; i < num_rows; i++)
	{
		for(int j = 0; j < num_cols; j++)
			mat[i*num_cols + j] = rand() % 4 + 1;
	}

	//init vector
	for (int i = 0; i < num_cols; i++)
	{
		vec[i] = rand() % 3 + 1;
	}


	//conduct matrix vector multiplication
	mat_vec_multiply(mat, vec, res, num_rows, num_cols);






	int *mat_d, *vec_d, *res_d, *res_gpu;
	H_ERR(hipMalloc ((void **) &mat_d, sizeof (int) * num_rows * num_cols));
	H_ERR(hipMalloc ((void **) &vec_d, sizeof (int) * num_cols));
	H_ERR(hipMalloc ((void **) &res_d, sizeof (int) * num_rows));
	res_gpu = (int *)malloc(sizeof(int) * num_rows);


	H_ERR(hipMemcpy (mat_d, mat, sizeof (int) * num_rows * num_cols, hipMemcpyHostToDevice));
	H_ERR(hipMemcpy (vec_d, vec, sizeof (int) * num_cols, hipMemcpyHostToDevice));

    gpu_mat_vec_multiply_shared <<< 256, 256 >>> (mat_d, vec_d, res_d, num_rows, num_cols);
	H_ERR(hipMemcpy (res_gpu, res_d, sizeof (int) * num_rows, hipMemcpyDeviceToHost));
	assert(memcmp(res_gpu, res, sizeof(int) * num_rows) == 0);
	std::cout<<"Shared Succeed !\n";
	
    
    gpu_mat_vec_multiply <<< 256, 256 >>> (mat_d, vec_d, res_d, num_rows, num_cols);
    H_ERR(hipMemcpy (res_gpu, res_d, sizeof (int) * num_rows, hipMemcpyDeviceToHost));
	assert(memcmp(res_gpu, res, sizeof(int) * num_rows) == 0);
	std::cout<<"Global Succeed !\n";

	return 0;
}
