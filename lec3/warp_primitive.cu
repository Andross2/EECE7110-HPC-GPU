
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <iostream>
#include <bitset>

__global__ void
warp_instruction (int *a, int *any, int *all, int *ballot, int *popc, int len)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	assert(blockDim.x * gridDim.x == 32);
    any[tid] = __any(a[tid]);
    all[tid] = __all(a[tid]);
    ballot[tid] = __ballot(a[tid]);
    popc[tid] = __popc(ballot[tid]);
}


    int
main (int args, char **argv)
{
    int len = 32;
    int *a = new int[len];
    int *any = new int[len];
    int *all = new int[len];
    int *ballot = new int[len];
    int *popc = new int[len];

    std::cout<<"Input data: \n";
    for (int i = 0; i < len; i++)
    {
        a[i] = rand () % 10;
        std::cout<<a[i]<<" ";
    }
    std::cout<<"\n";

    int *a_d, *any_d, *all_d, *ballot_d, *popc_d;

    hipMalloc ((void **) &a_d, sizeof (int) * len);
    hipMalloc ((void **) &any_d, sizeof (int) * len);
    hipMalloc ((void **) &all_d, sizeof (int) * len);
    hipMalloc ((void **) &ballot_d, sizeof (int) * len);
    hipMalloc ((void **) &popc_d, sizeof (int) * len);

    hipMemcpy (a_d, a, sizeof (int) * len, hipMemcpyHostToDevice);
    warp_instruction <<< 1, 32 >>> (a_d, any_d, all_d, ballot_d, popc_d, len);

    hipMemcpy (any, any_d, sizeof (int) * len, hipMemcpyDeviceToHost);
    hipMemcpy (all, all_d, sizeof (int) * len, hipMemcpyDeviceToHost);
    hipMemcpy (ballot, ballot_d, sizeof (int) * len, hipMemcpyDeviceToHost);
    hipMemcpy (popc, popc_d, sizeof (int) * len, hipMemcpyDeviceToHost);

    std::cout<<"any       all          ballot          popc:\n";
    for(int i = 0; i < len; i ++)
    std::cout<<"Thread "<<i<<": "<<any[i]<<" "<<all[i]<<" "
            <<std::bitset<32>(ballot[i])<<" "<<popc[i]<<"\n";
    return 0;
}
