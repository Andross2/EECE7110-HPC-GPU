
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <iostream>
#include <bitset>

__global__ void
block_reduction (int *a, int len)
{
    __shared__ int smem[256];
	assert(blockDim.x <= 256);


    smem[threadIdx.x] = threadIdx.x;

    __syncthreads();
    for (int i = blockDim.x/2; i > 0; i = i/2)
    {
        if(threadIdx.x < i)
        {
            int temp = smem[threadIdx.x] + smem[threadIdx.x + i];
            smem[threadIdx.x] = temp;
        }
        __syncthreads();

    }

    a[threadIdx.x] = smem[0];
}


    int
main (int args, char **argv)
{
    int len = 256;
    int *a = new int[len];

   int *a_d;

    hipMalloc ((void **) &a_d, sizeof (int) * len);

    block_reduction <<< 1, len >>> (a_d, len);

    hipMemcpy (a, a_d, sizeof (int) * len, hipMemcpyDeviceToHost);

    std::cout<<"Block id sum: "<< a[0]<<"\n";
    
    int sum = 0;
    for (int i = 0; i < 256; i ++)
        sum += i;

    std::cout<<"Correct result: "<<sum<<"\n";
    return 0;
}
